#include "hip/hip_runtime.h"
﻿#include <SDL.h>
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <cmath>

#undef main


const int SCREEN_WIDTH = 1280;
const int SCREEN_HEIGHT = 720;

SDL_Window* window;
SDL_Surface* screenSurface;

std::vector<std::vector<int>> world_map = {
        {1, 0, 0, 0},
        {0, 0, 0, 0},
        {0, 0, 0, 0},
        {0, 0, 0, 0}
};

float p_speler[] = {3, 3};
std::vector<double> r_speler = { -1.0 / std::sqrt(2), -1.0 / std::sqrt(2) };
std::vector<double> r_cameravlak = { 1.0 / std::sqrt(2), -1.0 / std::sqrt(2) };



void setupWindow() {
    if (SDL_Init(SDL_INIT_VIDEO) < 0) {
        std::cerr << "SDL could not initialize! SDL_Error: " << SDL_GetError() << std::endl;
    }

    window = SDL_CreateWindow("SDL Tutorial", SDL_WINDOWPOS_UNDEFINED, SDL_WINDOWPOS_UNDEFINED,
        SCREEN_WIDTH, SCREEN_HEIGHT, SDL_WINDOW_SHOWN);
    if (window == nullptr) {
        std::cerr << "Window could not be created! SDL_Error: " << SDL_GetError() << std::endl;
    }

    screenSurface = SDL_GetWindowSurface(window);
    SDL_FillRect(screenSurface, nullptr, SDL_MapRGB(screenSurface->format, 0xFF, 0xFF, 0xFF));
}

void drawSquare(int x, float length) {
    int squareX = x;
    int squareY = (SCREEN_HEIGHT - length) / 2;

    SDL_Rect squareRect = { squareX, squareY, 1, length };
    SDL_FillRect(screenSurface, &squareRect, SDL_MapRGB(screenSurface->format, 0x00, 0x00, 0xFF));
}

std::vector<double> get_rStraal(int column) {
    double d_camera = 1.0;
    std::vector<double> r_straal = { 0, 0 };
    r_cameravlak = { r_speler[1], -r_speler[0] };
    r_straal[0] = (d_camera * r_speler[0]) + ((2 * (column / static_cast<double>(SCREEN_WIDTH)) - 1) * r_cameravlak[0]);
    r_straal[1] = (d_camera * r_speler[1]) + ((2 * (column / static_cast<double>(SCREEN_WIDTH)) - 1) * r_cameravlak[1]);
    double length = std::sqrt(r_straal[0] * r_straal[0] + r_straal[1] * r_straal[1]);
    r_straal[0] /= length;
    r_straal[1] /= length;
    return r_straal;
}

std::vector<float> raycast(std::vector<double> r_straal) {
    float delta_v = 0.0;
    float delta_h = 0.0;
    float d_horizontaal = 0.0;
    float d_verticaal = 0.0;

    if (r_straal[0] == 0) {
        delta_v = 1e30;
    }
    else {
        delta_v = 1 / std::abs(r_straal[0]);
    }
    if (r_straal[1] == 0) {
        delta_h = 1e30;
    }
    else {
        delta_h = 1 / std::abs(r_straal[1]);
    }

    if (r_straal[1] < 0) {
        d_horizontaal = (p_speler[1] - std::floor(p_speler[1])) * delta_h;
    }
    else {
        d_horizontaal = (1 - p_speler[1] + std::floor(p_speler[1])) * delta_h;
    }

    if (r_straal[0] < 0) {
        d_verticaal = (p_speler[0] - std::floor(p_speler[0])) * delta_v;
    }
    else {
        d_verticaal = (1 - p_speler[0] + std::floor(p_speler[0])) * delta_v;
    }

    std::vector<double> i_coordinaat = { 0.0, 0.0 };
    bool hit = false;
    int intersectie = 1;
    int mapX = 0;
    int mapY = 0;
    float d_muur = 100;

    while (!hit) {
        if (d_horizontaal <= d_verticaal) {
            i_coordinaat[0] = p_speler[0] + d_horizontaal * r_straal[0];
            i_coordinaat[1] = round(p_speler[1] + d_horizontaal * r_straal[1]);
            d_horizontaal += delta_h;
            intersectie = 1;

            // Calculate the cell we want to check
            if (r_straal[1] < 0) {
                mapY = static_cast<int>(std::floor(i_coordinaat[1] - 1));
                mapX = static_cast<int>(std::floor(i_coordinaat[0]));
            }
            else { // Check cell ABOVE
                mapY = static_cast<int>(std::floor(i_coordinaat[1]));
                mapX = static_cast<int>(std::floor(i_coordinaat[0]));
            }
        }
        else {
            i_coordinaat[0] = round(p_speler[0] + d_verticaal * r_straal[0]);
            i_coordinaat[1] = round(p_speler[1] + d_verticaal * r_straal[1]);
            d_verticaal += delta_v;
            intersectie = 0;

            // Calculate the cell we want to check
            if (r_straal[0] < 0) { // Check cell LEFT
                mapY = static_cast<int>(std::floor(i_coordinaat[1]));
                mapX = static_cast<int>(std::floor(i_coordinaat[0] - 1));
            }
            else { // Check cell RIGHT
                mapY = static_cast<int>(std::floor(i_coordinaat[1]));
                mapX = static_cast<int>(std::floor(i_coordinaat[0]));
            }
        }

        // Check if the point we're going to check is within the map
        if (mapX > (world_map.size() - 1) || mapX < 0 || mapY >(world_map[0].size() - 1) || mapY < 0) {
            d_muur = 100;
            intersectie = 0;
            hit = true;
        }
        else {
            if (world_map[mapY][mapX] > 0) {
                d_muur = std::sqrt(((i_coordinaat[0] - p_speler[0]) * (i_coordinaat[0] - p_speler[0])) + ((i_coordinaat[1] - p_speler[1]) * (i_coordinaat[1] - p_speler[1])));
                hit = true;
            }
        }
    }

    return { d_muur, static_cast<float>(intersectie) };
}

void renderColumn(int column, float d_muur) {
    float length = ((2 * SCREEN_WIDTH) / 3) / d_muur;
    drawSquare(column, length);
}



int main(int argc, char* args[]) {
    
    setupWindow();

    float i = 0;
    bool quit = false;
    SDL_Event e;

    // Main loop
    while (!quit) {
        SDL_FillRect(screenSurface, nullptr, SDL_MapRGB(screenSurface->format, 0xFF, 0xFF, 0xFF));

        while (SDL_PollEvent(&e) != 0) {
            if (e.type == SDL_QUIT) {
                quit = true;
            }
        }

        for (int column = 0; column < SCREEN_WIDTH + 1; column++) {
            std::vector<double> r_straal = get_rStraal(column);
            std::vector<float> dMuur_Intersectie = raycast(r_straal);
            renderColumn(column, dMuur_Intersectie[0]);
        }

        SDL_UpdateWindowSurface(window);
    }

    SDL_DestroyWindow(window);
    SDL_Quit();

    return 0;
}
