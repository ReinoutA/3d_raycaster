
#include <hip/hip_runtime.h>
﻿/*
#include <SDL.h>
#include <iostream>
#include <vector>
#include <cmath>
#include <thread>
#include <string>
#include <SDL_image.h>
#include <stdio.h>
#include <sstream>
#include <fstream>
#undef main


const int SCREEN_WIDTH = 1280;
const int SCREEN_HEIGHT = 720;
const int IMG_SIZE = 512;
SDL_Window* window;
SDL_Surface* screenSurface;
Uint32* img;

int frame_rate = 0;
double elapsed_time = 0.0;
int world_map[15][15];
std::vector<double> fps_history;
int world_map_len = 15;
float p_speler[] = { 3, 3 };
float r_straal[] = { 1.0 / std::sqrt(2), -1.0 / std::sqrt(2) };
float r_speler[] = { 1 / sqrt(2) , -1 / sqrt(2) };
float r_cameravlak[] = { -1 / sqrt(2) , -1 / sqrt(2) };

void initializeWorldMap() {
    FILE* file = fopen("world_map.txt", "r");
    if (file == NULL) {
        printf("Kon het bestand niet openen.");
        return;
    }

    for (int i = 0; i < 15; i++) {
        for (int j = 0; j < 15; j++) {
            fscanf(file, "%d", &world_map[i][j]);
        }
    }
    fclose(file);
}

void setupWindow() {
    if (SDL_Init(SDL_INIT_VIDEO) < 0) {
        std::cerr << "SDL could not initialize! SDL_Error: " << SDL_GetError() << std::endl;
    }

    window = SDL_CreateWindow("SDL Raycaster", SDL_WINDOWPOS_UNDEFINED, SDL_WINDOWPOS_UNDEFINED, SCREEN_WIDTH, SCREEN_HEIGHT, SDL_WINDOW_SHOWN);
    if (window == nullptr) {
        std::cerr << "Window could not be created! SDL_Error: " << SDL_GetError() << std::endl;
    }

    screenSurface = SDL_GetWindowSurface(window);
    //SDL_FillRect(screenSurface, nullptr, SDL_MapRGB(screenSurface->format, 0xFF, 0xFF, 0xFF));
}

Uint32* loadImage(const char* file) {
    SDL_Surface* img = IMG_Load(file);
    img = SDL_ConvertSurfaceFormat(img, SDL_PIXELFORMAT_RGBA32, 0);
    return static_cast<Uint32*>(img->pixels);
}

void getImage(const char* file) {
    img = loadImage(file);
    if (img == NULL) {
        std::cout << "img was nullptr";
    }
}

float get_r_straal_x(int column) {
    return r_speler[0] + (2 * (column / static_cast<double>(SCREEN_WIDTH)) - 1) * r_cameravlak[0];
}

float get_r_straal_y(int column) {
    return r_speler[1] + (2 * (column / static_cast<double>(SCREEN_WIDTH)) - 1) * r_cameravlak[1];
}

void renderImageKolom(int kolom, float d_muur, int intersectie, float i_x, float i_y) {
    // Bereken de hoogte van het te renderen deel van de afbeelding op basis van de muurafstand
    int img_height_screen = static_cast<int>((SCREEN_HEIGHT / d_muur));
    float img_scale = (float)IMG_SIZE / img_height_screen;
    int render_y_begin = (SCREEN_HEIGHT - img_height_screen) / 2;
    float img_y;
    if (render_y_begin >= 0) {
        img_y = 0.0;
    }
    else {
        img_y = -render_y_begin;
    }

    // Bepaal de x-positie op de afbeelding op basis van de intersectie
    int img_x;
    if (intersectie == 1)
        img_x = (int)(IMG_SIZE * (i_x - std::floor(i_x)));
    else
        img_x = (int)(IMG_SIZE * (i_y - std::floor(i_y)));

    // Render elke pixel van de afbeelding op de kolom op het scherm
    for (int screen_y = std::max(render_y_begin, 0); screen_y < std::min(render_y_begin + img_height_screen, SCREEN_HEIGHT); ++screen_y, ++img_y) {
        // Bereken de index van de pixel in de afbeelding
        int img_idx = (int)(img_y * img_scale) * IMG_SIZE + img_x;
        // Bereken de index op het scherm
        int screen_idx = kolom + SCREEN_WIDTH * screen_y;
        // Kopieer de pixelwaarde van de afbeelding naar het scherm
        ((Uint32*)screenSurface->pixels)[screen_idx] = img[img_idx];
    }
}

void raycast(int column) {
    float delta_v = 0.0;
    float delta_h = 0.0;
    float d_horizontaal = 0.0;
    float d_verticaal = 0.0;

    // Bereken de coördinaten van r_cameravlak
    float r_straal_x = get_r_straal_x(column);
    float r_straal_y = get_r_straal_y(column);

    delta_v = (r_straal_x == 0) ? 1e30 : 1 / std::abs(r_straal_x);
    delta_h = (r_straal_y == 0) ? 1e30 : 1 / std::abs(r_straal_y);

    // Bereken d_horizontaal en d_verticaal
    d_horizontaal = (r_straal_y < 0) ? (p_speler[1] - std::floor(p_speler[1])) * delta_h : (1 - p_speler[1] + std::floor(p_speler[1])) * delta_h;
    d_verticaal = (r_straal_x < 0) ? (p_speler[0] - std::floor(p_speler[0])) * delta_v : (1 - p_speler[0] + std::floor(p_speler[0])) * delta_v;

    bool hit = false;
    int intersectie = 1;
    int mapX = 0;
    int mapY = 0;
    float d_muur = 100;
    float i_x, i_y;

    while (!hit) {
        if (d_verticaal >= d_horizontaal) {
            i_x = r_straal_x * d_horizontaal + p_speler[0];
            i_y = r_straal_y * d_horizontaal + p_speler[1];

            if (mapX > (world_map_len - 1) || mapX < 0 || mapY >(world_map_len - 1) || mapY < 0) {
                d_muur = 100;
                hit = true;
            }
            else {
                if (world_map[static_cast<int>(std::floor(i_x))][static_cast<int>(std::round(i_y) + ((r_straal_y < 0) ? -1.0 : 0.0))] > 0) {
                    d_muur = d_horizontaal * (r_straal_x * r_speler[0] + r_straal_y * r_speler[1]);
                    hit = true;
                }
            }
            mapX++;
            d_horizontaal += delta_h;
        }
        else {
            i_x = r_straal_x * d_verticaal + p_speler[0];
            i_y = r_straal_y * d_verticaal + p_speler[1];

            if (mapX > (world_map_len - 1) || mapX < 0 || mapY >(world_map_len - 1) || mapY < 0) {
                d_muur = 100;
                intersectie = 0;
                hit = true;
            }
            else {
                if (world_map[static_cast<int>(std::round(i_x) + ((r_straal_x < 0) ? -1.0 : 0.0))][static_cast<int>(std::floor(i_y))] > 0) {
                    d_muur = d_verticaal * (r_straal_x * r_speler[0] + r_straal_y * r_speler[1]);
                    intersectie = 0;
                    hit = true;
                }
            }
            mapY++;
            d_verticaal += delta_v;
        }

    }
    renderImageKolom(column, d_muur, intersectie, i_x, i_y);
}

// Functie om spelerbeweging te verwerken
void handleMovement(SDL_Event& e) {
    const Uint8* currentKeyStates = SDL_GetKeyboardState(nullptr);
    double speed = 0.01;
    double angularSpeed = 0.02; // Angulaire snelheid (rotatie per frame)

    // Check vooruit
    if (currentKeyStates[SDL_SCANCODE_W]) {
        p_speler[0] += r_speler[0] * speed;
        p_speler[1] += r_speler[1] * speed;
    }
    // Check achteruit
    if (currentKeyStates[SDL_SCANCODE_S]) {
        p_speler[0] -= r_speler[0] * speed;
        p_speler[1] -= r_speler[1] * speed;
    }
    // Check links
    if (currentKeyStates[SDL_SCANCODE_A]) {
        // Beweeg de speler zijwaarts naar links
        p_speler[0] -= r_speler[1] * speed;
        p_speler[1] += r_speler[0] * speed;
    }
    // Check rechts
    if (currentKeyStates[SDL_SCANCODE_D]) {
        // Beweeg de speler zijwaarts naar rechts
        p_speler[0] += r_speler[1] * speed;
        p_speler[1] -= r_speler[0] * speed;
    }

    // Check links (rotatie tegen de klok in)
    if (currentKeyStates[SDL_SCANCODE_F]) {
        double cosA = std::cos(-angularSpeed);
        double sinA = std::sin(-angularSpeed);
        double x = r_speler[0];
        r_speler[0] = cosA * x - sinA * r_speler[1];
        r_speler[1] = sinA * x + cosA * r_speler[1];

        // Bereken nieuwe richting voor r_cameravlak
        float new_r_cameravlak_x = cosA * r_cameravlak[0] - sinA * r_cameravlak[1];
        float new_r_cameravlak_y = sinA * r_cameravlak[0] + cosA * r_cameravlak[1];
        r_cameravlak[0] = new_r_cameravlak_x;
        r_cameravlak[1] = new_r_cameravlak_y;
    }
    // Check rechts (rotatie met de klok mee)
    if (currentKeyStates[SDL_SCANCODE_E]) {
        double cosA = std::cos(angularSpeed);
        double sinA = std::sin(angularSpeed);
        double x = r_speler[0];
        r_speler[0] = cosA * x - sinA * r_speler[1];
        r_speler[1] = sinA * x + cosA * r_speler[1];

        // Bereken nieuwe richting voor r_cameravlak
        float new_r_cameravlak_x = cosA * r_cameravlak[0] - sinA * r_cameravlak[1];
        float new_r_cameravlak_y = sinA * r_cameravlak[0] + cosA * r_cameravlak[1];
        r_cameravlak[0] = new_r_cameravlak_x;
        r_cameravlak[1] = new_r_cameravlak_y;
    }
}


void calculateAndSetFPSTitle(double deltaTime) {
    frame_rate++;
    elapsed_time += deltaTime;

    if (elapsed_time >= 1.0) {
        float frame_rate_per_sec = static_cast<double>(frame_rate) / elapsed_time;
        fps_history.push_back(frame_rate_per_sec); // Voeg de huidige fps toe aan de geschiedenis
        if (fps_history.size() > 1) { // Houd slechts de fps van de afgelopen 10 seconden bij
            fps_history.erase(fps_history.begin());
        }

        // Bereken gemiddelde FPS van de afgelopen 10 seconden
        double sum = 0.0;
        for (double fps : fps_history) {
            sum += fps;
        }
        double average_fps = sum / fps_history.size();

        // Open het bestand in append-modus en schrijf het gemiddelde FPS
        std::ofstream outfile("gemiddelde_fps.txt", std::ios::app);
        if (outfile.is_open()) {
            outfile << average_fps << std::endl;
            outfile.close();
        }
        else {
            std::cerr << "Kon gemiddelde_fps.txt niet openen voor schrijven!" << std::endl;
        }

        // Stel de venstertitel in met de huidige FPS
        std::stringstream stream;
        stream << "Frame Rate: " << static_cast<int>(frame_rate_per_sec);
        SDL_SetWindowTitle(window, stream.str().c_str());

        // Reset frame_rate en elapsed_time voor de volgende meting
        frame_rate = 0;
        elapsed_time = 0.0;
    }
}

int main(int argc, char* args[]) {

    setupWindow();

    getImage("muur.png");
    initializeWorldMap();
    bool quit = false;
    SDL_Event e;

    auto start_time = std::chrono::high_resolution_clock::now();
    while (!quit) {
        while (SDL_PollEvent(&e) != 0) {
            if (e.type == SDL_QUIT) {
                quit = true;
            }
        }

        handleMovement(e);

        for (int pixel_idx = 0; pixel_idx < SCREEN_WIDTH * SCREEN_HEIGHT; ++pixel_idx) {
            ((Uint32*)screenSurface->pixels)[pixel_idx] = 0xFFFFFFFF;
        }

        for (int column = 0; column <= SCREEN_WIDTH - 1; column++) {
            raycast(column);
        }

        SDL_UpdateWindowSurface(window);

        auto end_time = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> deltaTime = end_time - start_time;
        start_time = end_time;

        calculateAndSetFPSTitle(deltaTime.count());
    }
    SDL_DestroyWindow(window);
    SDL_Quit();

    return 0;
}
*/