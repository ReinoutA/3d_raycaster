#include "hip/hip_runtime.h"
﻿#include <SDL.h>
#include <iostream>
#include <vector>
#include <thread>
#include <string>
#include <SDL_image.h>
#include <stdio.h>
#include <sstream>
#include <hip/hip_runtime.h>
#include <fstream>
#undef main

const int SCREEN_WIDTH = 1280;
const int SCREEN_HEIGHT = 720;
const int IMG_SIZE = 512;

const int BLOCK_SIZE = 512;
const int NUM_BLOCKS = 3;

SDL_Window* window;
SDL_Surface* screenSurface;
Uint32* img;

int frame_rate = 0;
double elapsed_time = 0.0;
__constant__ int world_map[15][15];
std::vector<double> fps_history;
const int world_map_len = 15;
float p_speler[] = { 3, 3 };
float r_straal[] = { 1.0 / std::sqrt(2), -1.0 / std::sqrt(2) };
float r_speler[] = { 1 / sqrt(2) , -1 / sqrt(2) };
float r_cameravlak[] = { -1 / sqrt(2) , -1 / sqrt(2) };

void initializeWorldMap() {
    FILE* file = fopen("world_map.txt", "r");
    if (file == NULL) {
        printf("Kon het bestand niet openen.");
        return;
    }

    int temp[world_map_len][world_map_len];
    for (int i = 0; i < world_map_len; i++) {
        for (int j = 0; j < world_map_len; j++) {
            fscanf(file, "%d", &temp[i][j]);
        }
    }
    fclose(file);

    hipMemcpyToSymbol(HIP_SYMBOL(world_map), temp, sizeof(int) * world_map_len * world_map_len);
}



void setupWindow() {
    if (SDL_Init(SDL_INIT_VIDEO) < 0) {
        std::cerr << "SDL could not initialize! SDL_Error: " << SDL_GetError() << std::endl;
    }

    window = SDL_CreateWindow("SDL Raycaster", SDL_WINDOWPOS_UNDEFINED, SDL_WINDOWPOS_UNDEFINED, SCREEN_WIDTH, SCREEN_HEIGHT, SDL_WINDOW_SHOWN);
    if (window == nullptr) {
        std::cerr << "Window could not be created! SDL_Error: " << SDL_GetError() << std::endl;
    }

    screenSurface = SDL_GetWindowSurface(window);
    //SDL_FillRect(screenSurface, nullptr, SDL_MapRGB(screenSurface->format, 0xFF, 0xFF, 0xFF));
}

Uint32* loadImage(const char* file) {
    SDL_Surface* img = IMG_Load(file);
    img = SDL_ConvertSurfaceFormat(img, SDL_PIXELFORMAT_RGBA32, 0);
    return static_cast<Uint32*>(img->pixels);
}

void getImage(const char* file) {
    img = loadImage(file);
    if (img == NULL) {
        std::cout << "img was nullptr";
    }
}

__device__ float get_r_straal_x(int column, float* d_r_speler, float* d_r_cameravlak) {
    return d_r_speler[0] + (2 * (column / static_cast<double>(SCREEN_WIDTH)) - 1) * d_r_cameravlak[0];
}

__device__ float get_r_straal_y(int column, float* d_r_speler, float* d_r_cameravlak) {
    return d_r_speler[1] + (2 * (column / static_cast<double>(SCREEN_WIDTH)) - 1) * d_r_cameravlak[1];
}

__device__ inline int maximum(int a, int b) {
    return (a > b) ? a : b;
}

__device__ inline int minimum(int a, int b) {
    return (a < b) ? a : b;
}


__device__ void renderImageKolom(int kolom, float d_muur, int intersectie, float i_x, float i_y, Uint32* screen_gpu, int SCREEN_WIDTH, int SCREEN_HEIGHT, Uint32* img_gpu) {
    // Bereken de hoogte van het te renderen deel van de afbeelding op basis van de muurafstand
    int img_height_screen = static_cast<int>((SCREEN_HEIGHT / d_muur));
    float img_scale = (float)IMG_SIZE / img_height_screen;
    int render_y_begin = (SCREEN_HEIGHT - img_height_screen) / 2;
    float img_y;
    if (render_y_begin >= 0) {
        img_y = 0.0;
    }
    else {
        img_y = -render_y_begin;
    }

    // Bepaal de x-positie op de afbeelding op basis van de intersectie
    int img_x;
    if (intersectie == 1)
        img_x = (int)(IMG_SIZE * (i_x - std::floor(i_x)));
    else
        img_x = (int)(IMG_SIZE * (i_y - std::floor(i_y)));

    // Render elke pixel van de afbeelding op de kolom op het scherm
    for (int screen_y = maximum(render_y_begin, 0); screen_y < minimum(render_y_begin + img_height_screen, SCREEN_HEIGHT); ++screen_y, ++img_y) {
        // Bereken de index van de pixel in de afbeelding
        int img_idx = (int)(img_y * img_scale) * IMG_SIZE + img_x;
        // Bereken de index op het scherm
        int screen_idx = kolom + SCREEN_WIDTH * screen_y;
        // Kopieer de pixelwaarde van de afbeelding naar het scherm
        screen_gpu[screen_idx] = img_gpu[img_idx];
    }
}

__global__ void raycast_kernel_coalesced(float* p_speler, float* r_speler, Uint32* screen_gpu, int SCREEN_WIDTH, int SCREEN_HEIGHT, Uint32* img_gpu, float* d_p_speler, float* d_r_speler, float* d_r_cameravlak) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int totalThreads = gridDim.x * blockDim.x;

    int elementsPerThread = (SCREEN_WIDTH + totalThreads - 1) / totalThreads;
    int startColumn = idx * elementsPerThread;
    int endColumn = startColumn + elementsPerThread;

    for (int column = startColumn; column < endColumn; ++column) {
        if (column >= SCREEN_WIDTH) break;
        for (int row = 0; row < SCREEN_HEIGHT; row++) {
            int pixel_idx = column + row * SCREEN_WIDTH;
            screen_gpu[pixel_idx] = 0xFFFFFFFF;
        }

        float delta_v = 0.0;
        float delta_h = 0.0;
        float d_horizontaal = 0.0;
        float d_verticaal = 0.0;

        // Bereken de coördinaten van r_cameravlak
        float r_straal_x = get_r_straal_x(column, d_r_speler, d_r_cameravlak);
        float r_straal_y = get_r_straal_y(column, d_r_speler, d_r_cameravlak);

        delta_v = (r_straal_x == 0) ? 1e30 : 1 / std::abs(r_straal_x);
        delta_h = (r_straal_y == 0) ? 1e30 : 1 / std::abs(r_straal_y);

        // Bereken d_horizontaal en d_verticaal
        d_horizontaal = (r_straal_y < 0) ? (d_p_speler[1] - std::floor(d_p_speler[1])) * delta_h : (1 - d_p_speler[1] + std::floor(d_p_speler[1])) * delta_h;
        d_verticaal = (r_straal_x < 0) ? (d_p_speler[0] - std::floor(d_p_speler[0])) * delta_v : (1 - d_p_speler[0] + std::floor(d_p_speler[0])) * delta_v;

        bool hit = false;
        int intersectie = 1;
        int mapX = 0;
        int mapY = 0;
        float d_muur = 100;
        float i_x, i_y;

        while (!hit) {
            if (d_verticaal >= d_horizontaal) {
                i_x = r_straal_x * d_horizontaal + d_p_speler[0];
                i_y = r_straal_y * d_horizontaal + d_p_speler[1];

                if (mapX > (world_map_len - 1) || mapX < 0 || mapY >(world_map_len - 1) || mapY < 0) {
                    d_muur = 100;
                    hit = true;
                }
                else {
                    if (world_map[static_cast<int>(std::floor(i_x))][static_cast<int>(std::round(i_y) + ((r_straal_y < 0) ? -1.0 : 0.0))] > 0) {
                        d_muur = d_horizontaal * (r_straal_x * d_r_speler[0] + r_straal_y * d_r_speler[1]);
                        hit = true;
                    }
                }
                mapX++;
                d_horizontaal += delta_h;
            }
            else {
                i_x = r_straal_x * d_verticaal + p_speler[0];
                i_y = r_straal_y * d_verticaal + p_speler[1];

                if (mapX > (world_map_len - 1) || mapX < 0 || mapY >(world_map_len - 1) || mapY < 0) {
                    d_muur = 100;
                    intersectie = 0;
                    hit = true;
                }
                else {
                    if (world_map[static_cast<int>(std::round(i_x) + ((r_straal_x < 0) ? -1.0 : 0.0))][static_cast<int>(std::floor(i_y))] > 0) {
                        d_muur = d_verticaal * (r_straal_x * d_r_speler[0] + r_straal_y * d_r_speler[1]);
                        intersectie = 0;
                        hit = true;
                    }
                }
                mapY++;
                d_verticaal += delta_v;
            }

        }
        renderImageKolom(column, d_muur, intersectie, i_x, i_y, screen_gpu, SCREEN_WIDTH, SCREEN_HEIGHT, img_gpu);
    }
}

__global__ void raycast_kernel(float* p_speler, float* r_speler, Uint32* screen_gpu, int SCREEN_WIDTH, int SCREEN_HEIGHT, Uint32* img_gpu, float* d_p_speler, float* d_r_speler, float* d_r_cameravlak) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int threadCount = gridDim.x * blockDim.x;
    

    for (int column = idx; column < SCREEN_WIDTH; column += threadCount) {

        for (int row = 0; row < SCREEN_HEIGHT; row++) {
            int pixel_idx = column + row * SCREEN_WIDTH;
            screen_gpu[pixel_idx] = 0xFFFFFFFF;
        }

        float delta_v = 0.0;
        float delta_h = 0.0;
        float d_horizontaal = 0.0;
        float d_verticaal = 0.0;

        // Bereken de coördinaten van r_cameravlak
        float r_straal_x = get_r_straal_x(column, d_r_speler, d_r_cameravlak);
        float r_straal_y = get_r_straal_y(column, d_r_speler, d_r_cameravlak);

        delta_v = (r_straal_x == 0) ? 1e30 : 1 / std::abs(r_straal_x);
        delta_h = (r_straal_y == 0) ? 1e30 : 1 / std::abs(r_straal_y);

        // Bereken d_horizontaal en d_verticaal
        d_horizontaal = (r_straal_y < 0) ? (d_p_speler[1] - std::floor(d_p_speler[1])) * delta_h : (1 - d_p_speler[1] + std::floor(d_p_speler[1])) * delta_h;
        d_verticaal = (r_straal_x < 0) ? (d_p_speler[0] - std::floor(d_p_speler[0])) * delta_v : (1 - d_p_speler[0] + std::floor(d_p_speler[0])) * delta_v;

        bool hit = false;
        int intersectie = 1;
        int mapX = 0;
        int mapY = 0;
        float d_muur = 100;
        float i_x, i_y;

        while (!hit) {
            if (d_verticaal >= d_horizontaal) {
                i_x = r_straal_x * d_horizontaal + d_p_speler[0];
                i_y = r_straal_y * d_horizontaal + d_p_speler[1];

                if (mapX > (world_map_len - 1) || mapX < 0 || mapY >(world_map_len - 1) || mapY < 0) {
                    d_muur = 100;
                    hit = true;
                }
                else {
                    if (world_map[static_cast<int>(std::floor(i_x))][static_cast<int>(std::round(i_y) + ((r_straal_y < 0) ? -1.0 : 0.0))] > 0) {
                        d_muur = d_horizontaal * (r_straal_x * d_r_speler[0] + r_straal_y * d_r_speler[1]);
                        hit = true;
                    }
                }
                mapX++;
                d_horizontaal += delta_h;
            }
            else {
                i_x = r_straal_x * d_verticaal + p_speler[0];
                i_y = r_straal_y * d_verticaal + p_speler[1];

                if (mapX > (world_map_len - 1) || mapX < 0 || mapY >(world_map_len - 1) || mapY < 0) {
                    d_muur = 100;
                    intersectie = 0;
                    hit = true;
                }
                else {
                    if (world_map[static_cast<int>(std::round(i_x) + ((r_straal_x < 0) ? -1.0 : 0.0))][static_cast<int>(std::floor(i_y))] > 0) {
                        d_muur = d_verticaal * (r_straal_x * d_r_speler[0] + r_straal_y * d_r_speler[1]);
                        intersectie = 0;
                        hit = true;
                    }
                }
                mapY++;
                d_verticaal += delta_v;
            }

        }
        renderImageKolom(column, d_muur, intersectie, i_x, i_y, screen_gpu, SCREEN_WIDTH, SCREEN_HEIGHT, img_gpu);
    }
}


// Functie om spelerbeweging te verwerken
void handleMovement(SDL_Event& e) {
    const Uint8* currentKeyStates = SDL_GetKeyboardState(nullptr);
    double speed = 0.01;
    double angularSpeed = 0.02; // Angulaire snelheid (rotatie per frame)

    // Check vooruit
    if (currentKeyStates[SDL_SCANCODE_W]) {
        p_speler[0] += r_speler[0] * speed;
        p_speler[1] += r_speler[1] * speed;
    }
    // Check achteruit
    if (currentKeyStates[SDL_SCANCODE_S]) {
        p_speler[0] -= r_speler[0] * speed;
        p_speler[1] -= r_speler[1] * speed;
    }
    // Check links
    if (currentKeyStates[SDL_SCANCODE_A]) {
        // Beweeg de speler zijwaarts naar links
        p_speler[0] -= r_speler[1] * speed;
        p_speler[1] += r_speler[0] * speed;
    }
    // Check rechts
    if (currentKeyStates[SDL_SCANCODE_D]) {
        // Beweeg de speler zijwaarts naar rechts
        p_speler[0] += r_speler[1] * speed;
        p_speler[1] -= r_speler[0] * speed;
    }

    // Check links (rotatie tegen de klok in)
    if (currentKeyStates[SDL_SCANCODE_F]) {
        double cosA = std::cos(-angularSpeed);
        double sinA = std::sin(-angularSpeed);
        double x = r_speler[0];
        r_speler[0] = cosA * x - sinA * r_speler[1];
        r_speler[1] = sinA * x + cosA * r_speler[1];

        // Bereken nieuwe richting voor r_cameravlak
        float new_r_cameravlak_x = cosA * r_cameravlak[0] - sinA * r_cameravlak[1];
        float new_r_cameravlak_y = sinA * r_cameravlak[0] + cosA * r_cameravlak[1];
        r_cameravlak[0] = new_r_cameravlak_x;
        r_cameravlak[1] = new_r_cameravlak_y;
    }
    // Check rechts (rotatie met de klok mee)
    if (currentKeyStates[SDL_SCANCODE_E]) {
        double cosA = std::cos(angularSpeed);
        double sinA = std::sin(angularSpeed);
        double x = r_speler[0];
        r_speler[0] = cosA * x - sinA * r_speler[1];
        r_speler[1] = sinA * x + cosA * r_speler[1];

        // Bereken nieuwe richting voor r_cameravlak
        float new_r_cameravlak_x = cosA * r_cameravlak[0] - sinA * r_cameravlak[1];
        float new_r_cameravlak_y = sinA * r_cameravlak[0] + cosA * r_cameravlak[1];
        r_cameravlak[0] = new_r_cameravlak_x;
        r_cameravlak[1] = new_r_cameravlak_y;
    }
}


void calculateAndSetFPSTitle(double deltaTime) {
    frame_rate++;
    elapsed_time += deltaTime;

    if (elapsed_time >= 10.0) {
        float frame_rate_per_sec = static_cast<double>(frame_rate) / elapsed_time;
        fps_history.push_back(frame_rate_per_sec); // Voeg de huidige fps toe aan de geschiedenis
        if (fps_history.size() > 10) { // Houd slechts de fps van de afgelopen 10 seconden bij
            fps_history.erase(fps_history.begin());
        }

        // Bereken gemiddelde FPS van de afgelopen 10 seconden
        double sum = 0.0;
        for (double fps : fps_history) {
            sum += fps;
        }
        double average_fps = sum / fps_history.size();

        // Open het bestand in append-modus en schrijf het gemiddelde FPS
        std::ofstream outfile("gemiddelde_fps.txt", std::ios::app);
        if (outfile.is_open()) {
            outfile << average_fps << std::endl;
            outfile.close();
        }
        else {
            std::cerr << "Kon gemiddelde_fps.txt niet openen voor schrijven" << std::endl;
        }

        // Stel de venstertitel in met de huidige FPS
        std::stringstream stream;
        stream << "Frame Rate: " << static_cast<int>(frame_rate_per_sec);
        SDL_SetWindowTitle(window, stream.str().c_str());

        // Reset frame_rate en elapsed_time voor de volgende meting
        frame_rate = 0;
        elapsed_time = 0.0;
    }
}

std::chrono::high_resolution_clock::time_point last_blocksize_update = std::chrono::high_resolution_clock::now();
int block_size = 500;
bool blocksize_written = false; // Om bij te houden of blocksize al is weggeschreven
void updateBlockSize() {
    auto current_time = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed_time = current_time - last_blocksize_update;
    if (elapsed_time.count() >= 3.0) {
        block_size += 1; // Incrementeer de blocksize
        last_blocksize_update = current_time; // Update de tijd van de laatste blocksize-update
        blocksize_written = false; // Reset de flag zodat er opnieuw geschreven kan worden voor de nieuwe blocksize
    }
}

// Functie om FPS en blocksize naar een CSV-bestand te schrijven
void writeFPSAndBlockSizeToCSV(double fps) {
    if (!blocksize_written) { // Schrijf alleen als blocksize nog niet is weggeschreven
        std::ofstream csv_file("fps_blocksize.csv", std::ios::app);
        if (csv_file.is_open()) {
            csv_file << fps << "," << block_size << "\n";
            csv_file.close();
            blocksize_written = true; // Update de flag om aan te geven dat blocksize is weggeschreven
        }
        else {
            std::cerr << "Unable to open CSV file for writing!" << std::endl;
        }
    }
}

int main(int argc, char* args[]) {

    setupWindow();

    getImage("muur.png");
    initializeWorldMap();
    bool quit = false;
    SDL_Event e;

    // GPU
    float* d_p_speler;
    hipMalloc((void**)&d_p_speler, sizeof(float) * 2);
    float* d_r_speler;
    hipMalloc((void**)&d_r_speler, sizeof(float) * 2);
    float* d_r_cameravlak;
    hipMalloc((void**)&d_r_cameravlak, sizeof(float) * 2);
    Uint32* img_gpu;
    hipMalloc((void**)&img_gpu, sizeof(Uint32) * IMG_SIZE * IMG_SIZE);
    hipMemcpy(img_gpu, img, sizeof(Uint32) * IMG_SIZE * IMG_SIZE, hipMemcpyHostToDevice);
    Uint32* screen_gpu;
    hipMalloc((void**)&screen_gpu, sizeof(Uint32) * SCREEN_WIDTH * SCREEN_HEIGHT);
    hipMemcpy(screen_gpu, screenSurface->pixels, sizeof(Uint32) * SCREEN_WIDTH * SCREEN_HEIGHT, hipMemcpyHostToDevice);
    

    auto start_time = std::chrono::high_resolution_clock::now();


    while (!quit) {
        while (SDL_PollEvent(&e) != 0) {
            if (e.type == SDL_QUIT) {
                quit = true;
            }
        }
        updateBlockSize();

        handleMovement(e);

        // Copy data from host to device
        hipMemcpy(d_p_speler, p_speler, sizeof(float) * 2, hipMemcpyHostToDevice);
        hipMemcpy(d_r_speler, r_speler, sizeof(float) * 2, hipMemcpyHostToDevice);
        hipMemcpy(d_r_cameravlak, r_cameravlak, sizeof(float) * 2, hipMemcpyHostToDevice);

        //raycast_kernel_coalesced << <NUM_BLOCKS, BLOCK_SIZE >> > (d_p_speler, d_r_speler, screen_gpu, SCREEN_WIDTH, SCREEN_HEIGHT, img_gpu, d_p_speler, d_r_speler, d_r_cameravlak);
        raycast_kernel << <NUM_BLOCKS, block_size >> > (d_p_speler, d_r_speler, screen_gpu, SCREEN_WIDTH, SCREEN_HEIGHT, img_gpu, d_p_speler, d_r_speler, d_r_cameravlak);

        // Copy the updated screen buffer back to screenSurface->pixels
        hipMemcpy(screenSurface->pixels, screen_gpu, sizeof(Uint32) * SCREEN_WIDTH * SCREEN_HEIGHT, hipMemcpyDeviceToHost);

        // Update the window surface
        SDL_UpdateWindowSurface(window);


        auto end_time = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> deltaTime = end_time - start_time;
        start_time = end_time;
        calculateAndSetFPSTitle(deltaTime.count());

        double fps = 1.0 / deltaTime.count();
        writeFPSAndBlockSizeToCSV(fps);
    }
    hipFree(screen_gpu);
    hipFree(d_r_speler);
    hipFree(d_p_speler);
    hipFree(img_gpu);
    SDL_DestroyWindow(window);
    SDL_Quit();

    return 0;
}
